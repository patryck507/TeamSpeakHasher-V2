#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <vector>
#include <string>
#include <stdint.h>
#include <cstdlib>

#define CUDA_CHECK(expr)                                                         \
    do {                                                                        \
        hipError_t err = (expr);                                               \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error: " << hipGetErrorString(err)             \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl;\
            std::exit(1);                                                       \
        }                                                                       \
    } while (0)

// Use constant memory for the SHA1 round constants for faster access
__constant__ uint32_t k[4] = {0x5a827999, 0x6ed9eba1, 0x8f1bbcdc, 0xca62c1d6};

__device__ inline uint32_t ROTLEFT(uint32_t a, uint32_t b) {
    return (a << b) | (a >> (32 - b));
}

// Core SHA1 transform operating on a single 64 byte block
__device__ void sha1_transform(uint32_t state[5], const uint8_t data[64]) {
    uint32_t a, b, c, d, e, i, j, t, m[80];
    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | data[j + 3];
    for (; i < 80; ++i) {
        m[i] = m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16];
        m[i] = (m[i] << 1) | (m[i] >> 31);
    }

    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];

    for (i = 0; i < 20; ++i) {
        t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + k[0] + m[i];
        e = d; d = c; c = ROTLEFT(b, 30); b = a; a = t;
    }
    for (; i < 40; ++i) {
        t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + k[1] + m[i];
        e = d; d = c; c = ROTLEFT(b, 30); b = a; a = t;
    }
    for (; i < 60; ++i) {
        t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d)) + e + k[2] + m[i];
        e = d; d = c; c = ROTLEFT(b, 30); b = a; a = t;
    }
    for (; i < 80; ++i) {
        t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + k[3] + m[i];
        e = d; d = c; c = ROTLEFT(b, 30); b = a; a = t;
    }

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
}

// Kernel expects messages shorter than 56 bytes
__global__ void sha1_kernel(const uint8_t *input, size_t len, uint8_t *hash) {
    __shared__ uint8_t sdata[64]; // shared memory for faster access
    int tid = threadIdx.x;
    if (tid < len) sdata[tid] = input[tid];
    __syncthreads();

    if (tid == 0) {
        // pad remaining bytes
        sdata[len] = 0x80;
        for (size_t i = len + 1; i < 56; ++i) sdata[i] = 0;
        uint64_t bitlen = len * 8;
        for (int i = 0; i < 8; ++i) sdata[63 - i] = bitlen >> (8 * i);

        uint32_t state[5] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476, 0xC3D2E1F0};
        sha1_transform(state, sdata);
        for (int i = 0; i < 5; ++i) {
            hash[i * 4 + 0] = (state[i] >> 24) & 0xff;
            hash[i * 4 + 1] = (state[i] >> 16) & 0xff;
            hash[i * 4 + 2] = (state[i] >> 8) & 0xff;
            hash[i * 4 + 3] = state[i] & 0xff;
        }
    }
}

// Host helper wrapping the kernel
void sha1_cuda(const std::string &input, std::vector<uint8_t> &output) {
    uint8_t *d_in = nullptr, *d_out = nullptr;
    size_t len = input.size();
    CUDA_CHECK(hipMalloc(&d_in, len ? len : 1)); // allocate at least 1 byte
    CUDA_CHECK(hipMalloc(&d_out, 20));
    if (len)
        CUDA_CHECK(hipMemcpy(d_in, input.data(), len, hipMemcpyHostToDevice));
    sha1_kernel<<<1, 64>>>(d_in, len, d_out);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(output.data(), d_out, 20, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
}

std::string to_hex(const std::vector<uint8_t> &hash) {
    std::ostringstream oss;
    for (auto b : hash) {
        oss << std::hex << std::setw(2) << std::setfill('0') << static_cast<int>(b);
    }
    return oss.str();
}

bool run_test(const std::string &msg, const std::string &expected) {
    std::vector<uint8_t> out(20);
    sha1_cuda(msg, out);
    std::string hex = to_hex(out);
    bool ok = (hex == expected);
    std::cout << "SHA1('" << msg << "') = " << hex
              << (ok ? " [OK]" : " [FAIL]") << std::endl;
    return ok;
}

int main() {
    bool all_ok = true;
    all_ok &= run_test("", "da39a3ee5e6b4b0d3255bfef95601890afd80709");
    all_ok &= run_test("abc", "a9993e364706816aba3e25717850c26c9cd0d89d");
    return all_ok ? 0 : 1;
}
